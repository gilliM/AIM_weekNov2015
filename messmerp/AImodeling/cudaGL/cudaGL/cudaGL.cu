#include "hip/hip_runtime.h"
/*
 * Sample for OpenACC - OpenGL interoperability
 *
 * This sample runs a basic finite-difference solver of the 2D scalar wave equation. 
 *
 * usage: at program start, the sample launches a wave from the lower left corner into 
 *        the domain. Various hot keys are recognized:
 *        ' ' : restart the simulation
 *        'l' : add interior boundary condition ('logo')
 *        'n' : remove interior boundary condition ('no logo')
 *       'ESC': quit
 *       In addition, the user can interact with the scene via mouse (rotate and scale)
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h> 

//#include <inttypes.h>

#include <hip/hip_runtime.h>

// OpenGL Graphics includes
#include <GL/glew.h>
#include <GL/freeglut.h>

/*
#define __align__(x)
#define CUDARTAPI
#define __location__(a)
*/


#include "cuda_gl_interop.h"


#define REFRESH_DELAY     10 //ms

////////////////////////////////////////////////////////////////////////////////
// constants

// we want to be able to compile with a plain C compiler, so we have to 
// resort to cpp macros for defining constants
#define WINDOW_WIDTH 512
#define WINDOW_HEIGHT 512
const int window_width = WINDOW_WIDTH;
const int window_height = WINDOW_HEIGHT;

#define MESH_WIDTH 256
#define MESH_HEIGHT 256
const int mesh_width = MESH_WIDTH;
const int mesh_height = MESH_HEIGHT;

// our simulation mesh
float v[MESH_WIDTH * MESH_HEIGHT];       // velocity
float u[MESH_WIDTH * MESH_HEIGHT];       // amplitude


// physical units for grid spacing, time
const float dx2 = 1.0;
const float dy2 = 1.0;
const float dt  = 0.01;

// wave propertiess
#define G_C 100.
#define G_K (6.28 / 367.)

const float g_c = G_C;                   // phase velocity
const float g_k = G_K;                   // wave number.. just some nice number
const float g_w = G_K * G_C;             // wave frequency

// image dimension (for interior boundary condition)
#define IMG_X 248  
#define IMG_Y 189
const int img_x = IMG_X;
const int img_y = IMG_Y;

float img[IMG_X * IMG_Y];


// device variables, needed in CUDA version
float *d_u;
float *d_v;
float *d_img; 

typedef struct hipGraphicsResource cudaGraphicsResourceT;

// Vertex buffer
GLuint vbo;
cudaGraphicsResourceT* cuda_vbo_resource;
void *d_vbo_buffer = NULL;

// Normals buffer
GLuint normBuffer;
struct hipGraphicsResource* cuda_normBuffer_resource;

// Color buffer
GLuint colBuffer;
struct hipGraphicsResource* cuda_colBuffer_resource;

// index buffer for storing the indices of triangle strips
GLuint indexBuffer;

float g_fAnim = 0.0;      // global time
int g_fImg = 1;           // flag indicating logo to be displayd

// we artificially highlight the logo in the center. Color amplitude
// is increased over fade_in timesteps

//float fade_in = 300.;  looks good, too!
float fade_in = 500.;

// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;


int *pArgc = NULL;
char **pArgv = NULL;

#define MAX(a,b) ((a > b) ? a : b)

//__global__ void initKernel(const int mesh_width, const int mesh_height, float* d_v, float* d_u, float* d_ptr);


////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char **argv);
void runCuda();
void cleanup();

// GL functionality
void initGL(int *argc, char **argv);
void createVBO(GLuint *vbo, cudaGraphicsResourceT** cuda_res, int size, unsigned int vbo_res_flags);
void deleteVBO(GLuint *vbo);

void createMeshIndexBuffer(GLuint *id, int w, int h);

// rendering callbacks
void display();
void keyboard(unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);
void timerEvent(int value);

const char *sSDKsample = "OpenACC-OpenGL";


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    pArgc = &argc;
    pArgv = argv;

    printf("%s starting...\n", sSDKsample);
   
    runTest(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize GL
////////////////////////////////////////////////////////////////////////////////
void initGL(int *argc, char **argv)
{
    glutInit(argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(window_width, window_height);
    glutCreateWindow("OpenACC + OpenGL Interop");

    glutFullScreen();
    
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMotionFunc(motion);
    glutTimerFunc(REFRESH_DELAY, timerEvent,0);

    // initialize necessary OpenGL extensions
    glewInit();

    if (! glewIsSupported("GL_VERSION_2_0 "))
    {
        fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
        fflush(stderr);
    }

    // default initialization
    glClearColor(0.0, 0.0, 0.0, 1.0);
    //glDisable(GL_DEPTH_TEST);
    glEnable(GL_DEPTH_TEST);
    //glEnable(GL_LIGHTING); 
    glEnable(GL_LIGHT0);
    glEnable(GL_LIGHT1);

    // define material and surface properties
    GLfloat no_mat[] = { 0.0, 0.0, 0.0, 1.0 };
    GLfloat mat_ambient[] = { 0.2, 0.7, 0.1, 1.0 };
    GLfloat mat_ambient_color[] = { 0.2, 0.8, 0.2, 1.0 };
  // GLfloat mat_diffuse[] = { 0.1, 0.5, 0.8, 1.0 };
    GLfloat mat_diffuse[] = { 0.1, 0.5, 0.1, 1.0 };
    GLfloat mat_specular[] = { 1.0, 1.0, 1.0, 1.0 };
    GLfloat no_shininess[] = { 0.0 };
    GLfloat low_shininess[] = { 5.0 };
    GLfloat high_shininess[] = { 100.0 };
    GLfloat mat_emission[] = {0.3, 0.2, 0.2, 0.0};

    glTranslatef (1.25, 3.0, 0.0);
 //  glMaterialfv(GL_FRONT, GL_AMBIENT, no_mat);
    glMaterialfv(GL_FRONT, GL_AMBIENT, mat_ambient);
 
    glMaterialfv(GL_FRONT, GL_DIFFUSE, mat_diffuse);
    glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
    glMaterialfv(GL_FRONT, GL_SHININESS, high_shininess);
    glMaterialfv(GL_FRONT, GL_EMISSION, mat_emission);

    GLfloat light_ambient[] = { 0.0, 0.0, 0.0, 1.0 };
    GLfloat light_diffuse[] = { 1.0, 1.0, 1.0, 1.0 };
    GLfloat light_specular[] = { 1.0, 1.0, 1.0, 1.0 };
    GLfloat light_position[] = { 1.0, 1.0, 1.0, 0.0 };

    glLightfv(GL_LIGHT0, GL_AMBIENT, light_ambient);
    glLightfv(GL_LIGHT0, GL_DIFFUSE, light_diffuse);
    glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
    glLightfv(GL_LIGHT0, GL_POSITION, light_position);

    // viewport
    glViewport(0, 0, window_width, window_height);

    // projection
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0, (GLfloat)window_width / (GLfloat) window_height, 0.1, 10.0);

}

////////////////////////////////////////////////////////////////////////////////
//! Read the logo file and store it in the img array. The image is stored
//! in pgm format. We assume that the img array is already correctly sized
//! and no error checking is performed.
////////////////////////////////////////////////////////////////////////////////
void readImg(){
FILE* f;
char buf[1024];
int x,y, c;

  printf("reading logo\n");
  f = fopen("nvlogo_gray.pgm", "rt");
  fscanf(f, "%s", buf); // header 
  printf("line %s\n", buf); 
  fscanf(f, "%d %d", &x, &y);
  printf("%d %d\n", x, y);
  fscanf(f, "%d", &c);
  printf("%d\n", c);
  for(int y=0; y<img_y; y++){
    for(int x=0; x<img_x; x++){
      fscanf(f, "%d", &c);
      img[y*img_x + x]= c;
    }
  }
  printf("logo read\n");
  fclose(f);
}


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char **argv)
{
  initGL(&argc, argv);

  // register callbacks
  glutDisplayFunc(display);
  glutKeyboardFunc(keyboard);
  glutMouseFunc(mouse);
  glutMotionFunc(motion);

  // create VBO
  createVBO(&vbo, &cuda_vbo_resource, mesh_width*mesh_height*4*sizeof(float), 2);
  createVBO(&normBuffer, &cuda_normBuffer_resource, mesh_width*mesh_height*3*sizeof(float), 2);
  createVBO(&colBuffer, &cuda_colBuffer_resource, mesh_width*mesh_height*4*sizeof(float), 2);
  
  createMeshIndexBuffer(&indexBuffer, mesh_width, mesh_height);
   
  cudaGLSetGLDevice(0);

  for(int y=0; y<mesh_height; y++)
   for(int x=0; x<mesh_width; x++){ 
     u[(x+y*mesh_width)] = 0.0;
     v[(x+y*mesh_width)] = 0.0;
   }

  // load the logo
  readImg();

  // From here on the simulation variables live on the GPU

  hipMalloc(&d_u, MESH_WIDTH * MESH_HEIGHT * sizeof(float));
  hipMalloc(&d_v, MESH_WIDTH * MESH_HEIGHT * sizeof(float));
  hipMalloc(&d_img, IMG_X * IMG_Y * sizeof(float));

  hipMemcpy(d_u, u, MESH_WIDTH * MESH_HEIGHT * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_v, v, MESH_WIDTH * MESH_HEIGHT * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_img, img, IMG_X * IMG_Y * sizeof(float), hipMemcpyHostToDevice);

//#pragma acc data copy(u,v, img) /
//{
  glutMainLoop();
//}
  atexit(cleanup);
  return ;
}


////////////////////////////////////////////////////////////////////////////////
//! Create VBO
////////////////////////////////////////////////////////////////////////////////
void createVBO(GLuint *vbo, cudaGraphicsResourceT** cuda_res, int size, unsigned int vbo_res_flags)
{
    // create buffer object
    glGenBuffers(1, vbo);
    glBindBuffer(GL_ARRAY_BUFFER, *vbo);

    // initialize buffer object
    glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);

    glBindBuffer(GL_ARRAY_BUFFER, 0);

    // register this buffer object with CUDA
    hipGraphicsGLRegisterBuffer(cuda_res, *vbo, vbo_res_flags);
}


////////////////////////////////////////////////////////////////////////////////
//! Delete VBO
////////////////////////////////////////////////////////////////////////////////
void deleteVBO(GLuint *vbo)
{
    // unregister this buffer object with CUDA
    hipGraphicsUnregisterResource(cuda_vbo_resource);

    glBindBuffer(1, *vbo);
    glDeleteBuffers(1, vbo);

    *vbo = 0;
}

////////////////////////////////////////////////////////////////////////////////
//!  create index buffer for rendering the mesh via triangle strip
////////////////////////////////////////////////////////////////////////////////
void createMeshIndexBuffer(GLuint *id, int pw, int ph)
{   
    int w = pw;
    int h = ph;
  
    int size = (2 * w * h - h) * sizeof(GLuint);
    // create index buffer
    glGenBuffersARB(1, id);
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, *id);
    glBufferDataARB(GL_ELEMENT_ARRAY_BUFFER, size, 0, GL_STATIC_DRAW);

    // fill with indices for rendering mesh as triangle strips
    GLuint *indices = (GLuint *) glMapBuffer(GL_ELEMENT_ARRAY_BUFFER, GL_WRITE_ONLY);

    if (!indices)
    { 
        return;
    }

    int x, y;
    
     for (y=0; y<h-1; )
    {
        for (x=0; x<w; x++)
        {
            *indices++ = y*w+x;         
            *indices++ = (y+1)*w+x; 
        }
        y++;
        
        for (x = (w-2); x>=0; x--)
        {
            *indices++ = (y+1)*w+(x+1);    
            *indices++ = y*w+x; 
        }     
        y++;
    }

    glUnmapBuffer(GL_ELEMENT_ARRAY_BUFFER);
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, 0);
}


__global__ void initKernel(const int mesh_width, const int mesh_height, float* d_v, float* d_u, float* d_ptr)
{
    
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int tidy = blockIdx.y * blockDim.y + threadIdx.y;

    if(tidx < mesh_width && tidy < mesh_height) {
        d_v[tidx + tidy * mesh_width] = 0.0;
        d_u[tidx + tidy * mesh_width] = 0.0;
        float s = (float) tidx / (float) mesh_width;
        float t = (float) tidy / (float) mesh_height;
        d_ptr[(tidx + tidy*mesh_width) * 4    ] = 2.f * s - 1.f;
        d_ptr[(tidx + tidy*mesh_width) * 4 + 1] = 2.f * t - 1.f;
        d_ptr[(tidx + tidy*mesh_width) * 4 + 2] = 0.0;
        d_ptr[(tidx + tidy*mesh_width) * 4 + 3] = 1.0;
    }
};

__global__ void updateDisplacement(const int mesh_width, const int mesh_height, 
       const float* d_v, float* d_u, const float dt)
{
    
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int tidy = blockIdx.y * blockDim.y + threadIdx.y;

    if(tidx < mesh_width && tidy < mesh_height) {
        d_u[tidx + tidy * mesh_width] += 
             dt * d_v[tidx + tidy * mesh_width];
    }
};


__global__ void updateVelocity(const int mesh_width, const int mesh_height, 
       float* d_v, float* d_u, 
       const float dt, const float g_c, const float dx2, const float dy2)
{
    
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int tidy = blockIdx.y * blockDim.y + threadIdx.y;

    if(tidx < mesh_width-3 && tidy < mesh_height-3) {
        tidx++; 
        tidy++;

       d_v[tidx + tidy * mesh_width] += dt * g_c * (
              (d_u[(tidx + (tidy+1)*mesh_width)] 
       - 2.f * d_u[(tidx +   tidy  *mesh_width)] 
             + d_u[(tidx + (tidy-1)*mesh_width)]) / dx2 + 
              (d_u[(tidx+1 + tidy*mesh_width)] 
       - 2.f * d_u[(tidx   + tidy*mesh_width)] 
             + d_u[(tidx-1 + tidy*mesh_width)]) / dy2);
    }
};


__global__ void setBoundaryY(const int mesh_width, const int mesh_height, 
       float* d_u, float g_w, float g_k, float time) {
    
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int tidy = blockIdx.y * blockDim.y + threadIdx.y;

    if(tidx < mesh_width/4 && tidy < 2) {    
       d_u[tidx + tidy * mesh_width] = 
              cosf(g_w*time + g_k*tidx) * 
              sinf(g_w*time + g_k*tidy) ;
             // sinf(g_w*time + g_k*tidx) * 
             // cosf(g_w*time + g_k*tidy) ;

    }
};


__global__ void setBoundaryX(const int mesh_width, const int mesh_height, 
       float* d_u, float g_w, float g_k, float time) 
{
    
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int tidy = blockIdx.y * blockDim.y + threadIdx.y;

    if(tidx < 2 && tidy < mesh_height/4) {
       d_u[tidx + tidy * mesh_width] = 
              sinf(g_w*time + g_k*tidx) * 
              cosf(g_w*time + g_k*tidy) ;
    }
};


__global__ void setInteriorBoundary(const int mesh_width, float * d_v, float* d_u, 
     float * d_img, const int img_x, const int img_y, const int off_x, const int off_y, 
     float time, float fade_in) 
{
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int tidy = blockIdx.y * blockDim.y + threadIdx.y;

    if(tidx < img_x && tidy < img_y) {
        if(d_img[tidy * img_x + tidx] < 10.f) {
            d_u[(tidy + off_y) * mesh_width + (tidx + off_x)] = 0.;
            d_v[(tidy + off_y) * mesh_width + (tidx + off_x)] = 
                 (time < fade_in) ? sin(6.28*time/fade_in)*sin(6.28*time/fade_in) : 1.;
        } 
    }
}


__global__ void displacementToVBO(const int mesh_width, const int mesh_height, 
    const float *d_u, float *dptr) 
{
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int tidy = blockIdx.y * blockDim.y + threadIdx.y;

    if(tidx < mesh_width && tidy < mesh_height) {
       dptr[(tidy*mesh_width+tidx) *4 + 2] = d_u[tidy*mesh_width + tidx];
    }
}

__global__ void calcNormals(const int mesh_width, const int mesh_height, float *d_u, float* dptr_norm)
{
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int tidy = blockIdx.y * blockDim.y + threadIdx.y;

    if(tidx < mesh_width && tidy < mesh_height) {
        if(tidx < mesh_width-1 && tidy < mesh_height-1){
          dptr_norm[(tidy*mesh_width+tidx)*3+0] = d_u[(tidy*mesh_width+(tidx+1))]- d_u[(tidy*mesh_width+tidx)]; 
          dptr_norm[(tidy*mesh_width+tidx)*3+1] = d_u[((tidy+1)*mesh_width+tidx)]- d_u[(tidy*mesh_width+tidx)]; 
          dptr_norm[(tidy*mesh_width+tidx)*3+2] = 1.0; 
        } else {
          dptr_norm[(tidy*mesh_width+tidx)*3+0] = 0.0; 
          dptr_norm[(tidy*mesh_width+tidx)*3+1] = 0.0; 
          dptr_norm[(tidy*mesh_width+tidx)*3+2] = 1.0; 
        } 
    }
}

__global__ void calcColors(const int mesh_width, const int mesh_height, float *d_v, float* dptr_col)
{
 
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int tidy = blockIdx.y * blockDim.y + threadIdx.y;

    if(tidx < mesh_width && tidy < mesh_height) {
        if(tidx < mesh_width-1 && tidy < mesh_height-1) {
            dptr_col[(tidy*mesh_width+tidx)*4+0] = 0.1; 
            dptr_col[(tidy*mesh_width+tidx)*4+1] = (d_v[(tidy*mesh_width + tidx)] + 1.)/2.; 
            dptr_col[(tidy*mesh_width+tidx)*4+2] = (0.01*d_v[((tidy+1)*mesh_width+tidx)]+0.01)/2.;  
            dptr_col[(tidy*mesh_width+tidx)*4+3] = 1.0; 
        } else {
            dptr_col[(tidy*mesh_width+tidx)*4+0] = 0.1; 
            dptr_col[(tidy*mesh_width+tidx)*4+1] = 0.5; 
            dptr_col[(tidy*mesh_width+tidx)*4+2] = 0.0;  
            dptr_col[(tidy*mesh_width+tidx)*4+3] = 1.0; 
        }
    } 
}  




////////////////////////////////////////////////////////////////////////////////
//! Run the CUDA part of the computation
////////////////////////////////////////////////////////////////////////////////
void runCuda()
{
    // map OpenGL buffer objects for writing from CUDA
    float * __restrict__ dptr;
    hipGraphicsMapResources(1, &cuda_vbo_resource, 0);

    size_t num_bytes;
    hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes, cuda_vbo_resource);

    float * __restrict__ dptr_norm;
    hipGraphicsMapResources(1, &cuda_normBuffer_resource, 0);
    size_t num_bytes_norm;
    hipGraphicsResourceGetMappedPointer((void **)&dptr_norm, &num_bytes_norm, cuda_normBuffer_resource);

    float * __restrict__ dptr_col;
    
    hipGraphicsMapResources(1, &cuda_colBuffer_resource, 0);
    size_t num_bytes_col;
    hipGraphicsResourceGetMappedPointer((void **)&dptr_col, &num_bytes_col, cuda_colBuffer_resource);

    // just for convenience
    float time = g_fAnim;

    int nThreadsX = 16;
    int nThreadsY = 16;
    int nBlocksX = (mesh_width + nThreadsX - 1) / nThreadsX;
    int nBlocksY = (mesh_height+ nThreadsY - 1) / nThreadsY;
    dim3 grid(nBlocksX, nBlocksY);
    dim3 block(nThreadsX, nThreadsY);

    if(time==0.0f){
      // clear out the simulation fields
      initKernel<<<grid, block>>>(mesh_width, mesh_height, d_v, d_u, dptr);
     }

   updateDisplacement<<<grid, block>>>(mesh_width, mesh_height, d_v, d_u, dt);

   updateVelocity<<<grid, block>>>(mesh_width, mesh_height, d_v, d_u, dt, g_c, dx2, dy2);

   setBoundaryY<<<dim3((mesh_width+63)/64), dim3(64, 2)>>>(mesh_width, mesh_height, d_u, g_w, g_k, time);

   setBoundaryX<<<dim3((mesh_height+63)/64), dim3(2, 64)>>>(mesh_width, mesh_height, d_u, g_w, g_k, time);
 

   // if desired, set interior boundary condition   
   if( g_fImg == 1){
       
     // we want to offset the image a bit
     int off_x = 30;
     int off_y = 50;

     setInteriorBoundary<<<dim3((img_x+15)/16, (img_y+15)/16), block>>>(mesh_width, d_v, d_u, d_img, 
         img_x, img_y, off_x, off_y, time, fade_in);
   }


   displacementToVBO<<<grid, block>>>(mesh_width, mesh_height, d_u, dptr);

   calcNormals<<<grid, block>>>(mesh_width, mesh_height, d_u, dptr_norm);

   calcColors<<<grid, block>>>(mesh_width, mesh_height, d_v, dptr_col);
 

    // unmap buffer object
    hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0);
    hipGraphicsUnmapResources(1, &cuda_normBuffer_resource, 0);
    hipGraphicsUnmapResources(1, &cuda_colBuffer_resource, 0);
}



////////////////////////////////////////////////////////////////////////////////
//! Display callback
////////////////////////////////////////////////////////////////////////////////
void display()
{
    // run CUDA kernel to generate vertex positions
    runCuda();

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    // set view matrix
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    GLfloat lightpos[] = {0., 0., 0., 1.};
    
    glLightfv(GL_LIGHT0, GL_POSITION, lightpos);
   
    glTranslatef(0.0, 0.0, translate_z);
    glRotatef(rotate_x, 1.0, 0.0, 0.0);
    glRotatef(rotate_y, 0.0, 1.0, 0.0);
    
    glBindBuffer(GL_ARRAY_BUFFER, normBuffer);
    glNormalPointer(GL_FLOAT, 0, 0);
    glEnableClientState(GL_NORMAL_ARRAY);
    
    glBindBuffer(GL_ARRAY_BUFFER, colBuffer);
    glColorPointer(4, GL_FLOAT, 0, 0);
    glEnableClientState(GL_COLOR_ARRAY);

    // render from the vbo
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glVertexPointer(4, GL_FLOAT, 0, 0);
    glEnableClientState(GL_VERTEX_ARRAY);

    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, indexBuffer);

    // glPolygonMode(GL_FRONT_AND_BACK, wireFrame ? GL_LINE : GL_FILL);
    // glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
    glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
        
    glDrawElements(GL_TRIANGLE_STRIP, 2*(mesh_width-1)*(mesh_height-1) + mesh_width, GL_UNSIGNED_INT, 0);
       
    glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);

    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, 0);

    glDisableClientState(GL_COLOR_ARRAY);
    glDisableClientState(GL_VERTEX_ARRAY);
    glDisableClientState(GL_NORMAL_ARRAY);
 
    glutSwapBuffers();

    g_fAnim += dt;
}

void timerEvent(int value)
{
    glutPostRedisplay();
    glutTimerFunc(REFRESH_DELAY, timerEvent,0);
}

void cleanup()
{
}


////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard(unsigned char key, int x, int y)
{
    switch (key)
    {
        case (27) :
            hipDeviceReset();
            exit(EXIT_SUCCESS);
            break;
        case (' '):      // reset simulation
            g_fAnim = 0.0;
            break;

        case ('l'):      // turn on interior boundary condition
            g_fImg = 1;
            break;

        case ('n'):      // turn off interior boundary conditions
            g_fImg = 0;
 
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN)
    {
        mouse_buttons |= 1<<button;
    }
    else if (state == GLUT_UP)
    {
        mouse_buttons = 0;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

void motion(int x, int y)
{
    float dx, dy;
    dx = (float)(x - mouse_old_x);
    dy = (float)(y - mouse_old_y);

    if (mouse_buttons & 1)
    {
        rotate_x += dy * 0.2f;
        rotate_y += dx * 0.2f;
    }
    else if (mouse_buttons & 4)
    {
        translate_z += dy * 0.01f;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

